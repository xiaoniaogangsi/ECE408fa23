#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 8
#define MAX_MASK_WIDTH 7
#define MAX_CHANNEL_NUM 4
#define MAX_STRIDE 4
#define MAX_OUT_FM 16

#define MAX_SIN_WIDTH  ((TILE_WIDTH-1) * MAX_STRIDE + MAX_MASK_WIDTH)    //The maximum width of the shared input
#define MAX_SIN_CHSIZE  (MAX_SIN_WIDTH * MAX_SIN_WIDTH)            //The maximum size of each channel in the shared input 
#define MAX_SIN_SIZE  (MAX_CHANNEL_NUM * MAX_SIN_CHSIZE)            //The maximum size of the shared input
#define MAX_SMASK_CHSIZE  (MAX_MASK_WIDTH * MAX_MASK_WIDTH)          //The maximum size of each channel in the shared mask
#define MAX_SMASK_SIZE  (MAX_CHANNEL_NUM * MAX_SMASK_CHSIZE)        //The maximum size of each channel in the shared mask

__constant__ float Mask_c[MAX_OUT_FM * MAX_CHANNEL_NUM * MAX_MASK_WIDTH * MAX_MASK_WIDTH];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    /*Optimization 1: Tiled shared memory convolution
    Inside each block, each output element needs K*K elements from the input feature and from the mask,
    so there are many loads from global memory. We can copy elements from input and mask into shared memory.*/

    
    __shared__ float subTile_input[MAX_SIN_SIZE];
    // __shared__ float subTile_mask[MAX_SMASK_SIZE];

    int s_in_width = (TILE_WIDTH-1) * S + K;
    int s_in_chsize = s_in_width * s_in_width;      //The size of each channel in the shared input 
    int s_in_size = C * s_in_chsize;                //The size of the shared input

    int s_mask_chsize = K * K;              //The size of each channel in the shared mask
    int s_mask_size = C * s_mask_chsize;    //The size of each channel in the shared mask

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working
    const int H_grid = ceil(H_out / (1.0 * TILE_WIDTH));    //Number of tiles in height for one output feature map
    const int W_grid = ceil(W_out / (1.0 * TILE_WIDTH));    //Number of tiles in width for one output feature map

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    //out_4d(b, m, h_out, w_out)
    //in_4d(b, c, h, w)
    //mask_4d(m, c, p, q)
    #define shared_in_3d(i2, i1, i0) subTile_input[(i2) * (s_in_width * s_in_width) + (i1) * (s_in_width) + i0]
    // #define shared_mask_3d(i2, i1, i0) subTile_mask[(i2) * (K * K) + (i1) * (K) + i0]
    //shared_in_3d(c, h, w)
    //shared_mask_3d(c, p, q)
    #define const_mask_4d(i3, i2, i1, i0) Mask_c[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    //const_mask_4d(m, c, p, q)

    // Insert your GPU convolution kernel code here
    int m = blockIdx.x; //The index of the output feature map.
    //We need to recombine the linearized tiles to a feature map with (W_grid * H_grid) tiles.
    //Then use (h, w) to index each element in the recombined feature maps.
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y; //The height index of the output tile.
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x; //The width index of the output tile.
    int b = blockIdx.z; //The index of the image in the batch.

    //Copy the input feature map and mask into shared memory.
    int tile_index = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int tile_size = TILE_WIDTH * TILE_WIDTH;
    int h_base = (blockIdx.y / W_grid) * TILE_WIDTH;
    int w_base = (blockIdx.y % W_grid) * TILE_WIDTH;

    for (int index = tile_index; index < s_in_size; index += tile_size) {
        int c = index / s_in_chsize;
        int h_in = (index % s_in_chsize) / s_in_width;
        int w_in = (index % s_in_chsize) % s_in_width;
        if (h_base*S + h_in < H && w_base*S + w_in < W) {
            shared_in_3d(c, h_in, w_in) = in_4d(b, c, h_base*S + h_in, w_base*S + w_in);
        }
        else {
            shared_in_3d(c, h_in, w_in) = 0.0f;
        }
    }
    __syncthreads();

    // for (int index = tile_index; index < s_mask_size; index += tile_size) {
    //     int c = index / s_mask_chsize;
    //     int p = (index % s_mask_chsize) / K;
    //     int q = (index % s_mask_chsize) % K;
    //     shared_mask_3d(c, p, q) = mask_4d(m, c, p, q);
    // }
    // __syncthreads();

    if (h<H_out && w<W_out) {   //Check the boundary conditions of the output
        float acc = 0.0f;
        for (int c=0; c<C; c++) {   //Iterate on each channel
            for (int p=0; p<K; p++) {
                for (int q=0; q<K; q++) {   //Iterate on each mask element (there are K*K mask elements)
                    //Multiply the mask element with the corresponding input element and accumulate the result.
                    // if (h*S+p < H && w*S+q < W){    //Check the boundary condition of input, remember the stride S.
                    //     acc += in_4d(b, c, h*S+p, w*S+q) * mask_4d(m, c, p, q);
                    // }
                    if (threadIdx.y*S+p < s_in_width && threadIdx.x*S+q < s_in_width) {
                        // acc += shared_in_3d(c, threadIdx.y*S+p, threadIdx.x*S+q) * shared_mask_3d(c, p, q);
                        acc += shared_in_3d(c, threadIdx.y*S+p, threadIdx.x*S+q) * const_mask_4d(m, c, p, q);
                    }
                }
            }
        }
        out_4d(b, m, h, w) = acc;   //Store the result into output.
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d

    #undef shared_in_3d
    // #undef shared_mask_3d

    #undef const_mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
   
    hipMalloc((void **)device_output_ptr, B * M * H_out * W_out * sizeof(float));
    hipMalloc((void **)device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void **)device_mask_ptr, M * C * K * K * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Mask_c), host_mask, M * C * K * K * sizeof(float));
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;    //Output height for one output feature map
    const int W_out = (W - K)/S + 1;    //Output width for one output feature map
    const int H_grid = ceil(H_out / (1.0 * TILE_WIDTH));    //Number of tiles in height for one output feature map
    const int W_grid = ceil(W_out / (1.0 * TILE_WIDTH));    //Number of tiles in width for one output feature map
    const int Y = H_grid * W_grid;      //Number of tiles for one output feature map (linearized)

    dim3 dimGrid(M, Y, B);  //There are total B images in a batch. And for each image, there are M output feature map, each with Y tiles.
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);   //The dimension of a block (a tile)
    
    //---Debug---
    // std::cout<<"H_out: "<<H_out<<std::endl;
    // std::cout<<"W_out: "<<W_out<<std::endl;
    // std::cout<<"H_grid: "<<H_grid<<std::endl;
    // std::cout<<"W_grid: "<<W_grid<<std::endl;
    // std::cout<<"Y: "<<Y<<std::endl;
    // std::cout<<"dimGrid: x="<<dimGrid.x<<", y="<<dimGrid.y<<", z="<<dimGrid.z<<std::endl;
    // std::cout<<"dimBlock: x="<<dimBlock.x<<", y="<<dimBlock.y<<", z="<<dimBlock.z<<std::endl;
    //---Debug---

    //Launch the convolution forward kernel.
    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize();
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    hipMemcpy(host_output, device_output, B * M * H_out * W_out * sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
