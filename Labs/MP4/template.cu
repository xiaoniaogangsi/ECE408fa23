#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_WIDTH 4  //TILE_WIDTH is the same as BLOCK_WIDTH
//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];  //3*3*3=27 elements in total

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  //In each dimension, input sizes are (MASK_WIDTH-1) larger than the output sizes.
  int x_o = blockIdx.x * TILE_WIDTH + tx; //_o means in output coordinates
  int y_o = blockIdx.y * TILE_WIDTH + ty;
  int z_o = blockIdx.z * TILE_WIDTH + tz;
  int x_i = x_o - (MASK_WIDTH/2); //_i means in input coordinates
  int y_i = y_o - (MASK_WIDTH/2);
  int z_i = z_o - (MASK_WIDTH/2);

  //Allocate the shared memory for the block (tile)
  __shared__ float N_ds[TILE_WIDTH+(MASK_WIDTH-1)][TILE_WIDTH+(MASK_WIDTH-1)][TILE_WIDTH+(MASK_WIDTH-1)];
  
  //Load the shared memory N_ds, note that halos are also loaded.
  if ((x_i >= 0) && (x_i < x_size) && (y_i >= 0) && (y_i < y_size) && (z_i >= 0) && (z_i < z_size)){
    //If this thread takes values from valid global memory range:
    N_ds[tz][ty][tx] = input[z_i*y_size*x_size + y_i*x_size + x_i];
  }else{
    //If this thread takes values outside the balid global memory range:
    N_ds[tz][ty][tx] = 0.0f;
  }
  __syncthreads();  //To make sure N_ds is totally loaded

  //Perform the convolution between N_ds and Mc
  float Pvalue = 0.0f;
  if ((tx < TILE_WIDTH) && (ty < TILE_WIDTH) && (tz < TILE_WIDTH)) {
    for (int i=0; i<MASK_WIDTH; i++) {    //loop on z
      for (int j=0; j<MASK_WIDTH; j++){   //loop on y
        for (int k=0; k<MASK_WIDTH; k++){ //loop on x
          Pvalue += Mc[i][j][k] * N_ds[tz+i][ty+j][tx+k];
        }
      }
    }
    if ((x_o < x_size) && (y_o < y_size) && (z_o < z_size)){
      output[z_o*y_size*x_size + y_o*x_size + x_o] = Pvalue;
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void **) &deviceInput, (inputLength-3) * sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength-3) * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  //Use the address of hostInput[3] as the source address
  hipMemcpy(deviceInput, hostInput+3, (inputLength-3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mc), hostKernel, kernelLength * sizeof(float), 0, hipMemcpyHostToDevice);  //offset=0, kind=hipMemcpyHostToDevice
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimGrid(ceil(x_size/(1.0*TILE_WIDTH)), ceil(y_size/(1.0*TILE_WIDTH)), ceil(z_size/(1.0*TILE_WIDTH)));
  dim3 dimBlock(TILE_WIDTH + (MASK_WIDTH - 1), TILE_WIDTH + (MASK_WIDTH - 1), TILE_WIDTH + (MASK_WIDTH - 1));

  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput+3, deviceOutput, (inputLength-3) * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
